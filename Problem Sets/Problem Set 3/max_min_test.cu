#include "utils.h"

void min_or_max_driver(const float* const d_array, 
                       float* h_out, const size_t numElems,
                       bool is_max);

int main(int argc, char** argv) {
    int N = 1030;
    float h_array[N];
    float h_min, h_max;
    for (int i = 0; i < N; i++) 
        h_array[i] = i;
    float *d_array;

    checkCudaErrors(hipMalloc((void **) &d_array, N*sizeof(float)));
    checkCudaErrors(hipMemcpy(d_array, h_array, N, hipMemcpyDeviceToHost));

    min_or_max_driver(d_array, &h_min, N, false);
    min_or_max_driver(d_array, &h_max, N, true);

    checkCudaErrors(hipFree(d_array));
}